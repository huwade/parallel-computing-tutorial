#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matmul.h"
#include <iostream>
#define BLOCK_SIZE 32

// Matrix multiplication kernel called by MatMul()
__global__ void matrixMultiplyShared(Matrix A, Matrix B, Matrix C)
{

    float Cvalue = 0;
    // Thread row and column within Csub
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Shared memory used to store Asub and Bsub respectively
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    for (int m = 0; m < (A.column / BLOCK_SIZE); ++m)
    {
        As[threadIdx.y][threadIdx.x] = A.elements[(blockIdx.y * BLOCK_SIZE + threadIdx.y) * A.column + BLOCK_SIZE * m + threadIdx.x];
        Bs[threadIdx.y][threadIdx.x] = B.elements[(m * BLOCK_SIZE + threadIdx.y) * B.column + blockIdx.x * BLOCK_SIZE + threadIdx.x];

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    // Write Csub to device memory
    // Each thread writes one element
    C.elements[row * B.column + col] = Cvalue;
}

namespace matmul
{
    void MatmulOperator::mat_mul_cuda_shared(const Matrix &A, const Matrix &B, Matrix &C)
    {
        // Load A and B to device memory
        Matrix d_A;
        d_A.column = A.column;
        d_A.row = A.row;
        size_t size = A.column * A.row * sizeof(float);
        hipMalloc(&d_A.elements, size);
        hipMemcpy(d_A.elements, A.elements, size,
                   hipMemcpyHostToDevice);
        Matrix d_B;
        d_B.column = B.column;
        d_B.row = B.row;
        size = B.column * B.row * sizeof(float);
        hipMalloc(&d_B.elements, size);
        hipMemcpy(d_B.elements, B.elements, size,
                   hipMemcpyHostToDevice);
        // Allocate C in device memory
        Matrix d_C;
        d_C.column = C.column;
        d_C.row = C.row;
        size = C.column * C.row * sizeof(float);
        hipMalloc(&d_C.elements, size);

        // Invoke kernel
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(C.column / dimBlock.x, C.row / dimBlock.y);
        matrixMultiplyShared<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
        // Read C from device memory
        hipMemcpy(C.elements, d_C.elements, size,
                   hipMemcpyDeviceToHost);
        // Free device memory
        hipFree(d_A.elements);
        hipFree(d_B.elements);
        hipFree(d_C.elements);
    }
}