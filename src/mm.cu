
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <random>
#include <stdexcept>
#include <vector>

#define BLOCK_DIM 32

#define checkCuda(val) check((val), #val, __FILE__, __LINE__)

template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <typename T>
std::vector<T> create_rand_vector(size_t n)
{
    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(-256, 256);

    std::vector<T> vec(n);

    for (size_t i{0}; i < n; i++)
    {
        vec.at(i) = static_cast<T>(uniform_dist(e));
    }

    return vec;
}

template <typename T>
bool allclose(std::vector<T> const &vec_1, std::vector<T> const &vec_2,
              T const &abs_tol)
{
    if (vec_1.size() != vec_2.size())
        return false;

    for (size_t i{0}; i < vec_1.size(); i++)
    {
        if (std::abs(vec_1.at(i) - vec_2.at(i)) > abs_tol)
        {
            std::cout << vec_1.at(i) << " " << vec_2.at(i) << std::endl;
            return false;
        }
    }
    return true;
}

// mat_1: m x n
// mat_2: n x p
// mat_3: m x p
// C[i][j] += A[i][k] * B[k][j]
template <typename T>
void mm(T const *mat_1, T const *mat_2, T *mat_3, size_t m, size_t n, size_t p)
{
    for (size_t i{0}; i < m; i++)
    {
        for (size_t j{0}; j < p; j++)
        {
            T acc_sum{static_cast<T>(0)};
            for (size_t k{0}; k < n; k++)
            {
                acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
            }
            mat_3[i * p + j] = acc_sum;
        }
    }
}

template <typename T>
__global__ void mm_kernel(T const *mat_1, T const *mat_2, T *mat_3, size_t m,
                          size_t n, size_t p)
{
    // 2D block and 2D thread
    // Each thread computes one cell in mat_3.
    size_t C_row_idx{blockIdx.y * blockDim.y + threadIdx.y};
    size_t C_col_idx{blockIdx.x * blockDim.x + threadIdx.x};

    // Do not process outside the matrix.
    // Do not forget the equal sign!
    // mat_3: n x p
    if (C_row_idx < m && C_col_idx < p)
    {
        T acc_sum{static_cast<T>(0)};
        for (size_t k{0}; k < n; ++k)
        {
            acc_sum += mat_1[C_row_idx * n + k] * mat_2[k * p + C_col_idx];
        }
        mat_3[C_row_idx * p + C_col_idx] = acc_sum;
    }
}

template <typename T>
void mm_cuda(T const *mat_1, T const *mat_2, T *mat_3, size_t m, size_t n, size_t p)
{
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks_per_grid(1, 1);
    blocks_per_grid.x = std::ceil(static_cast<double>(p) /
                                  static_cast<double>(threads_per_block.x));

    blocks_per_grid.y = std::ceil(static_cast<double>(m) /
                                  static_cast<double>(threads_per_block.y));
    mm_kernel<<<blocks_per_grid, threads_per_block>>>(mat_1, mat_2, mat_3, m, n, p);
}

// mat_1: m x n
// mat_2: n x p
// mat_3: m x p
template <typename T>
bool random_test_mm_cuda(size_t m, size_t n, size_t p)
{
    std::vector<T> const mat_1_vec{create_rand_vector<T>(m * n)};
    std::vector<T> const mat_2_vec{create_rand_vector<T>(n * p)};
    // vector<int> vec(10); Creates a vector of size 10, all elements initialized to 0.
    // vector<int> vec{10}; Creates a vector with one element initialized to 10.
    std::vector<T> mat_3_vec(m * p);
    std::vector<T> mat_4_vec(m * p);
    T const *mat_1{mat_1_vec.data()};
    T const *mat_2{mat_2_vec.data()};
    T *mat_3{mat_3_vec.data()};
    T *mat_4{mat_4_vec.data()};

    mm(mat_1, mat_2, mat_3, m, n, p);

    T *d_mat_1, *d_mat_2, *d_mat_4;
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * mat_1_vec.size()));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * mat_2_vec.size()));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * mat_4_vec.size()));

    checkCuda(hipMemcpy(d_mat_1, mat_1, sizeof(T) * mat_1_vec.size(),
                         hipMemcpyHostToDevice));

    checkCuda(hipMemcpy(d_mat_2, mat_2, sizeof(T) * mat_2_vec.size(),
                         hipMemcpyHostToDevice));

    mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    hipDeviceSynchronize();
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }

    // Copy data from device to host.
    checkCuda(hipMemcpy(mat_4, d_mat_4, sizeof(T) * mat_4_vec.size(),
                         hipMemcpyDeviceToHost));

    // Free device buffer.
    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    return allclose<T>(mat_3_vec, mat_4_vec, 1e-4);
}

template <typename T>
bool random_multiple_test_mm_cuda(size_t num_tests)
{
    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(1, 256);

    size_t m{0}, n{0}, p{0};
    bool success{false};

    for (size_t i{0}; i < num_tests; i++)
    {
        m = static_cast<size_t>(uniform_dist(e));
        n = static_cast<size_t>(uniform_dist(e));
        p = static_cast<size_t>(uniform_dist(e));
        success = random_test_mm_cuda<T>(m, n, p);
        if (!success)
        {
            return false;
        }
    }
    return true;
}

template <typename T>
float measure_latency_mm_cuda(size_t m, size_t n, size_t p, size_t num_tests,
                              size_t num_warmups)
{
    hipEvent_t startEvent, stopEvent;
    float time{0.0f};

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    T *d_mat_1, *d_mat_2, *d_mat_4;

    // Allocate device buffer.
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * m * n));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * n * p));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * m * p));

    for (size_t i{0}; i < num_warmups; ++i)
    {
        mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    }

    checkCuda(hipEventRecord(startEvent, 0));
    for (size_t i{0}; i < num_tests; ++i)
    {
        mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    }
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
    checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));

    // Free device buffer.
    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    float latency{time / num_tests};

    return latency;
}

int main()
{
    constexpr size_t num_tests{10};

    assert(random_multiple_test_mm_cuda<int32_t>(num_tests));
    assert(random_multiple_test_mm_cuda<float>(num_tests));
    assert(random_multiple_test_mm_cuda<double>(num_tests));
    constexpr size_t num_measurement_tests{10};
    constexpr size_t num_measurement_warmups{10};
    size_t m{1024}, n{1024}, p{1024};

    float mm_cuda_int32_latency{measure_latency_mm_cuda<int32_t>(
        m, n, p, num_measurement_tests, num_measurement_warmups)};
    float mm_cuda_float_latency{measure_latency_mm_cuda<float>(
        m, n, p, num_measurement_tests, num_measurement_warmups)};
    float mm_cuda_double_latency{measure_latency_mm_cuda<double>(
        m, n, p, num_measurement_tests, num_measurement_warmups)};

    std::cout << "Matrix Multiplication CUDA Latency" << std::endl;
    std::cout << "m: " << m << " "
              << "n: " << n << " "
              << "p: " << p << std::endl;
    std::cout << "INT32: " << std::fixed << std::setprecision(5)
              << mm_cuda_int32_latency << " ms" << std::endl;
    std::cout << "FLOAT: " << std::fixed << std::setprecision(5)
              << mm_cuda_float_latency << " ms" << std::endl;
    std::cout << "DOUBLE: " << std::fixed << std::setprecision(5)
              << mm_cuda_double_latency << " ms" << std::endl;
}
